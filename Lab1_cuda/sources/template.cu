#include "hip/hip_runtime.h"
#include <gputk.h>

__global__ void vecAdd(float *in1, float *in2, float *out, int len) {
  //@@ Insert code to implement vector addition here
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < len) {
    out[idx] = in1[idx] + in2[idx]; 
  }
}

int main(int argc, char **argv) {
  gpuTKArg_t args;
  int inputLength;
  float *hostInput1;
  float *hostInput2;
  float *hostOutput;
  float *deviceInput1;
  float *deviceInput2;
  float *deviceOutput;

  args = gpuTKArg_read(argc, argv);

  gpuTKTime_start(Generic, "Importing data and creating memory on host");
  hostInput1 =
      (float *)gpuTKImport(gpuTKArg_getInputFile(args, 0), &inputLength);
  hostInput2 =
      (float *)gpuTKImport(gpuTKArg_getInputFile(args, 1), &inputLength);
  hostOutput = (float *)malloc(inputLength * sizeof(float));
  gpuTKTime_stop(Generic, "Importing data and creating memory on host");

  gpuTKLog(TRACE, "The input length is ", inputLength);

  gpuTKTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here
  hipMalloc((void**)&deviceInput1, sizeof(float) * inputLength);
  hipMalloc((void**)&deviceInput2, sizeof(float) * inputLength);
  hipMalloc((void**)&deviceOutput, sizeof(float) * inputLength);

  gpuTKTime_stop(GPU, "Allocating GPU memory.");

  gpuTKTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here
  hipMemcpy(deviceInput1, hostInput1, sizeof(float) * inputLength, hipMemcpyHostToDevice);
  hipMemcpy(deviceInput2, hostInput2, sizeof(float) * inputLength, hipMemcpyHostToDevice);

  gpuTKTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here
  int blockSize = 256;
  int gridSize = (inputLength + 255) / 256;
  gpuTKTime_start(Compute, "Performing CUDA computation");
  //@@ Launch the GPU Kernel here
  vecAdd<<<gridSize, blockSize>>>(deviceInput1, deviceInput2, deviceOutput, inputLength);

  hipDeviceSynchronize();
  gpuTKTime_stop(Compute, "Performing CUDA computation");

  gpuTKTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostOutput, deviceOutput, sizeof(float) * inputLength, hipMemcpyDeviceToHost);

  gpuTKTime_stop(Copy, "Copying output memory to the CPU");

  gpuTKTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here
  hipFree(deviceInput1);
  hipFree(deviceInput2);
  hipFree(deviceOutput);

  gpuTKTime_stop(GPU, "Freeing GPU Memory");

  gpuTKSolution(args, hostOutput, inputLength);

  free(hostInput1);
  free(hostInput2);
  free(hostOutput);

  return 0;
}
